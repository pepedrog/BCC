#include <stdio.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>


// Device global variables
__device__ double c_x_min;
__device__ double c_x_max;
__device__ double c_y_min;
__device__ double c_y_max;

__device__ double pixel_width;
__device__ double pixel_height;

__device__ int iteration_max = 200;

__device__ int image_size;
__device__ int image_buffer_size;

__device__ int num_threads;
__device__ int th_per_block;
__device__ int pixels_per_thread;

__device__ int gradient_size = 16;
__device__ int colors[17][3] = {
    {66, 30, 15},
    {25, 7, 26},
    {9, 1, 47},
    {4, 4, 73},
    {0, 7, 100},
    {12, 44, 138},
    {24, 82, 177},
    {57, 125, 209},
    {134, 181, 229},
    {211, 236, 248},
    {241, 233, 191},
    {248, 201, 95},
    {255, 170, 0},
    {204, 128, 0},
    {153, 87, 0},
    {106, 52, 3},
    {16, 16, 16},
};

// Host global variables
dim3 num_blocks, threads_per_block;
int num_blocks_x, th_per_block_x;
int num_blocks_y, th_per_block_y;
int host_image_buffer_size;
unsigned char* image_buffer_host;

int i_x_max;
int i_y_max;

int check (hipError_t& err, const char* msg) {
    if (err != hipSuccess) {
        printf ("%s", msg);
        printf (" | Error: %s\n", hipGetErrorString(err));
        return 1;
    }
    return 0;
}

void print_bad_arguments () {
    printf("usage: ./mandelbrot_seq c_x_min c_x_max c_y_min c_y_max"
    " image_size NUM_BLOCKS TH_PER_BLOCK \n");
    printf("examples with image_size = 11500:\n");
    printf("    Full Picture:         ./mandelbrot_cuda -2.5 1.5 -2.0 2.0 11500 4 64 \n");
    printf("    Seahorse Valley:      ./mandelbrot_cuda -0.8 -0.7 0.05 0.15 11500 4 64 \n");
    printf("    Elephant Valley:      ./mandelbrot_cuda 0.175 0.375 -0.1 0.1 11500 4 64 \n");
    printf("    Triple Spiral Valley: ./mandelbrot_cuda -0.188 -0.012 0.554 0.754 11500 4 64 \n");
}
// Get global variables from command line args
void init (int argc, char* argv[]) {
    // host variables
    double host_c_x_min, host_c_x_max;
    double host_c_y_min, host_c_y_max;
    int host_image_size;

    if (argc < 8) {
        print_bad_arguments();
        exit(0);
    }
    else {
        num_blocks_y = th_per_block_y = 1;

        sscanf(argv[1], "%lf", &host_c_x_min);
        sscanf(argv[2], "%lf", &host_c_x_max);
        sscanf(argv[3], "%lf", &host_c_y_min);
        sscanf(argv[4], "%lf", &host_c_y_max);
        sscanf(argv[5], "%d", &host_image_size);
        sscanf(argv[6], "%d", &num_blocks_x);
        if (argc == 8) {
            sscanf(argv[7], "%d", &th_per_block_x);
        }
        else if (argc == 10) {
            sscanf(argv[7], "%d", &num_blocks_y);
            sscanf(argv[8], "%d", &th_per_block_x);
            sscanf(argv[9], "%d", &th_per_block_y);
        }
        else {
            print_bad_arguments();
            exit(0);
        }

        host_image_buffer_size = host_image_size * host_image_size;

        int host_th_per_block = th_per_block_x * th_per_block_y;
        int host_num_threads = host_th_per_block * num_blocks_x * num_blocks_y;

        int host_pixels_per_thread = host_image_buffer_size / host_num_threads;

        i_x_max = host_image_size;
        i_y_max = host_image_size;
        double host_pixel_width = (host_c_x_max - host_c_x_min) / i_x_max;
        double host_pixel_height = (host_c_y_max - host_c_y_min) / i_y_max;
        // copy host variables to device
        hipError_t err = hipSuccess;
        hipMemcpyToSymbol(HIP_SYMBOL(c_x_min), &host_c_x_min, sizeof(double));
        hipMemcpyToSymbol(HIP_SYMBOL(c_x_max), &host_c_x_max, sizeof(double));
        hipMemcpyToSymbol(HIP_SYMBOL(c_y_min), &host_c_y_min, sizeof(double));
        hipMemcpyToSymbol(HIP_SYMBOL(c_y_max), &host_c_y_max, sizeof(double));
        hipMemcpyToSymbol(HIP_SYMBOL(image_size), &host_image_size, sizeof(int));
        hipMemcpyToSymbol(HIP_SYMBOL(num_threads), &host_num_threads, sizeof(int));
        hipMemcpyToSymbol(HIP_SYMBOL(th_per_block), &host_th_per_block, sizeof(int));
        hipMemcpyToSymbol(HIP_SYMBOL(pixel_width), &host_pixel_width, sizeof(double));
        hipMemcpyToSymbol(HIP_SYMBOL(pixel_height), &host_pixel_height, sizeof(double));
        hipMemcpyToSymbol(HIP_SYMBOL(pixels_per_thread), &host_pixels_per_thread, sizeof(int));
        hipMemcpyToSymbol(HIP_SYMBOL(image_buffer_size), &host_image_buffer_size, sizeof(int));
        err = hipGetLastError();
        if (check(err, "Failed to copy command line args to device"))
            exit(EXIT_FAILURE);
    };
};

__device__
void update_rgb_buffer(unsigned char* image_buffer_device, int iteration, int pix) {
    int color;

    if (iteration == iteration_max) {
        image_buffer_device[pix * 3 + 0] = colors[gradient_size][0];
        image_buffer_device[pix * 3 + 1] = colors[gradient_size][1];
        image_buffer_device[pix * 3 + 2] = colors[gradient_size][2];
    } else {
        color = iteration % gradient_size;
        image_buffer_device[pix * 3 + 0] = colors[color][0];
        image_buffer_device[pix * 3 + 1] = colors[color][1];
        image_buffer_device[pix * 3 + 2] = colors[color][2];
    };
};

__global__
void compute_mandelbrot(unsigned char* image_buffer_device) {

    double z_x;
    double z_y;
    double z_x_squared;
    double z_y_squared;
    double escape_radius_squared = 4;

    int iteration;
    int i_x;
    int i_y;

    double c_x;
    double c_y;

    // Calculates pixel where current thread will start its work
    int my_block = blockIdx.x + gridDim.x * blockIdx.y;
    int my_thread_in_block = threadIdx.x + blockDim.x * threadIdx.y;

    int my_thread = my_block * th_per_block + my_thread_in_block;

    /* what thread will process each pixel ?
     *
     * Example: image 5x5 -> buffer_size = 25
     * 3 blocks of 3 threads -> 9 threads
     *
     * 2 4 7 - -
     * 1 4 6 - -
     * 1 3 6 8 -
     * 0 3 5 8 -
     * 0 2 5 7 -
     *
     * and the remaining pixels we process separetedly,
     * each thread process its remaining pixel in the end
     *
     * 2 4 7 5 0
     * 1 4 6 6 1
     * 1 3 6 8 2
     * 0 3 5 8 3
     * 0 2 5 7 4
     */

     // Its easier to process by pixels instead of by row-collunm
    int pix = my_thread * pixels_per_thread;
    int end_pixel = pix + pixels_per_thread;
    int my_rem_pixel = image_buffer_size - my_thread - 1;

    while (pix <= my_rem_pixel) {
        i_y = pix / image_size;
        i_x = pix % image_size;

        c_y = c_y_min + i_y * pixel_height;
        if (fabs(c_y) < pixel_height / 2) {
            c_y = 0.0;
        };

        c_x = c_x_min + i_x * pixel_width;

        z_x = 0.0;
        z_y = 0.0;

        z_x_squared = 0.0;
        z_y_squared = 0.0;

        for (iteration = 0;
            iteration < iteration_max && \
            ((z_x_squared + z_y_squared) < escape_radius_squared);
            iteration++) {
            z_y = 2 * z_x * z_y + c_y;
            z_x = z_x_squared - z_y_squared + c_x;
            z_x_squared = z_x * z_x;
            z_y_squared = z_y * z_y;
        };
        
        update_rgb_buffer(image_buffer_device, iteration, pix);

        pix++;

        // Treat remaining pixel
        if (pix == end_pixel) {
            if (my_rem_pixel >= pix) pix = my_rem_pixel;
            else break;
        }
    }
}

void allocate_image_buffer(unsigned char** image_buffer_device, size_t size) {
    // Our buffer, instead of a matrix, will be a continuous array

    // Allocate host memory
    image_buffer_host = (unsigned char*)malloc(sizeof(unsigned char) * size);

    // Allocate device memory
    hipError_t err = hipSuccess;
    err = hipMalloc((void**)(image_buffer_device), size);

    // Test alloc success
    if (image_buffer_host == NULL) {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }
    if (check(err, "Failed to allocate device image buffer"))
        exit(EXIT_FAILURE);
};

void write_to_file() {
    FILE* file;
    const char* filename = "output.ppm";
    const char* comment = "# ";
    int max_color_component_value = 255;

    file = fopen(filename, "wb");

    fprintf(file, "P6\n %s\n %d\n %d\n %d\n", comment,
        i_x_max, i_y_max, max_color_component_value);

    for (int i = 0; i < host_image_buffer_size; i++) {
        fwrite(image_buffer_host + 3*i, 1, 3, file);
    };
    fclose(file);
};

int main(int argc, char* argv[]) {
    init(argc, argv);

    hipError_t err;
    int rgb_size = 3;
    size_t size = host_image_buffer_size * rgb_size;

    unsigned char* image_buffer_device;
    allocate_image_buffer(&image_buffer_device, size);

    // Launch compute_mandelbrot CUDA Kernel
    num_blocks = dim3(num_blocks_x, num_blocks_y);
    threads_per_block = dim3(th_per_block_x, th_per_block_y);
    compute_mandelbrot<<<num_blocks, threads_per_block>>>(image_buffer_device);
    hipDeviceSynchronize();
    err = hipGetLastError();
    if (check(err, "Failed to launch compute_mandelbrot kernel"))
        exit(EXIT_FAILURE);

    // Copy the device result vector in device memory to the host result vector
    // in host memory.
    err = hipMemcpy(image_buffer_host, image_buffer_device, size,
        hipMemcpyDeviceToHost);
    if (check(err, "Failed to copy vector from device to host"))
        exit(EXIT_FAILURE);

    // Free device global memory
    err = hipFree(image_buffer_device);
    if (check(err, "Failed to free device vector"))
        exit(EXIT_FAILURE);

    write_to_file();
    // Free host memory
    free(image_buffer_host);

    return 0;
}